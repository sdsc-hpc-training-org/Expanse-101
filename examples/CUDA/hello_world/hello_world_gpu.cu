// Cuda By Example - By Sanders and Kudrot
//
// Hello World Program in CUDA C
//
// Contains a function that is executed on the device (GPU)
//


#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void my_kernel(void){
 // nothing done here 
}

int main(void) {

  my_kernel<<<1,1>>>();
  printf("Hello World!\n");
  return 0;

}
